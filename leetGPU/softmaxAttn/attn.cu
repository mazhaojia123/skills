#include <hip/hip_runtime.h>
#include <float.h>

static __device__ float blockReduceSum(float *red, float v) {
    // red 是 reduction 用的共享内存
    // v 是当前线程所掌握的数据量
    unsigned int tId = threadIdx.x;
    red[tId] = v;
    __syncthreads();
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (tId < i) red[tId] += red[tId + i];
        __syncthreads();
    }
    
    return red[0];
}

__global__ void softmaxAttn(
    const float *Q,
    const float *K,
    const float *V, 
    float *O, 
    const int m, const int n, const int d)
{
    unsigned int bId = blockIdx.x;
    unsigned int tId = threadIdx.x;
    unsigned int TPB = blockDim.x;
    unsigned int BPG = gridDim.x;
    if (bId >= m) return;

    extern __shared__ float smem[];
    float *q_sh = smem;             // [d], 缓存Q[i,:]
    float *red = smem + d;          // [blockDim.x]
    float *scal = red + blockDim.x; // [1]

    float ro[16];
    for (int i = 0; i < 16; i++) ro[i] = 0.0f;

    float inv_sqrt_d = rsqrtf((float)d);

    const float *q0 = Q + bId * d;
    for (int k0 = 0; k0 < d; k0++) {
        q_sh[k0] = q0[k0];
    }

    __syncthreads();

    float row_max = -FLT_MAX;
    for (int col = 0; col < n; col++) {
        const float *k0 = K + col * d;
        float acc = 0.0f;
        for (unsigned int idx = tId; idx < d; idx += TPB) {
            acc += q_sh[idx] * k0[idx];
        }
        acc = blockReduceSum(red, acc);
        if (tId == 0) {
            float logit = acc * inv_sqrt_d;
            row_max = fmaxf(row_max, logit);
        }
    }

    if (tId == 0) scal[0] = row_max;
    __syncthreads();
    row_max = scal[0];

    float denom = 0.0f;
    for (int col = 0; col < n; col++) {
        const float *k0 = K + col * d;
        float acc = 0.0f;
        for (unsigned int idx = tId; idx < d; idx += TPB) {
            acc += q_sh[idx] * k0[idx];
        }
        acc = blockReduceSum(red, acc);
        if (tId == 0) {
            float logit = acc * inv_sqrt_d;
            float e = expf(logit - row_max);
            denom += e;
            scal[1] = e;
        }
        __syncthreads();
        float p = scal[1];
        const float *v0 = V + col * d;
        for (int i = tId; i < d; i += TPB) {
            ro[i / TPB] += p * v0[i];
        }
    }

    if (tId == 0) scal[2] = denom;
    __syncthreads();
    denom = scal[2];

    float *o0 = O + bId * d;
    for (int i = tId; i < d; i += TPB) {
        o0[i] = ro[i / TPB] / denom;
    }
}

// Q, K, V, output are device pointers
extern "C" void solve(const float* Q, const float* K, const float* V, float* output, int M, int N, int d) {
    // Q (Mxd), K (Nxd), V (Nxd)

    int threads = 1;
    while (threads < d && threads < 256) threads <<= 1; // 1,2,4,...,256
    dim3 grid(M);
    dim3 block(threads);

    size_t smemSize = (threads + d + 3) * sizeof(float);
    softmaxAttn<<<grid, block, smemSize>>>(Q, K, V, output, M, N, d);
    hipDeviceSynchronize();
}
