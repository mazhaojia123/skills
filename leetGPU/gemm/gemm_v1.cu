#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
using namespace nvcuda;

#define WM 16
#define WN 16
#define WK 16

__global__ void gemm_kernel(
    const half* A, const half* B, half* C, int M, int N, int K, float alpha, float beta)
{
    const int tileM = blockIdx.x * WM;
    const int tileN = blockIdx.y * WN;

    extern __shared__ unsigned char smemRaw[];
    half *sA = reinterpret_cast<half*>(smemRaw);        // WM * WK
    half *sB = sA + WM * WK;                            // WK * WN
    float *sC = reinterpret_cast<float*>(sB + WK * WN); // WM * WN

    const int laneId = threadIdx.x & 31;
    wmma::fragment<wmma::accumulator, WM, WN, WK, float> fragC;
    wmma::fill_fragment(fragC, 0.0f);

    for (int k0 = 0; k0 < K; k0 += WK) {
        // load A to sA
        for (int i = laneId; i < WM * WK; i += 32) {
            int c = i % WK;
            int r = i / WK;
            int gc = k0 + c;
            int gr = tileM + r;
            half rA = 0.0f;
            if (gc < K && gr < M) rA = A[gc + gr * K];
            sA[c + r * WK] = rA;
        }

        // load B to sB
        for (int i = laneId; i < WN * WK; i += 32) {
            int c = i % WN;
            int r = i / WN;
            int gc = c + tileN;
            int gr = r + k0;
            half rB = 0.0f;
            if (gc < N && gr <K) rB = B[gc + gr * N];
            sB[c + r * WN] = rB;
        }

        __syncthreads();

        wmma::fragment<wmma::matrix_a, WM, WN, WK, half, wmma::row_major> fragA;
        wmma::fragment<wmma::matrix_b, WM, WN, WK, half, wmma::row_major> fragB;

        wmma::load_matrix_sync(fragA, sA, WK);
        wmma::load_matrix_sync(fragB, sB, WN);

        wmma::mma_sync(fragC, fragA, fragB, fragC);

        __syncthreads();
    }

    wmma::store_matrix_sync(sC, fragC, WN, wmma::mem_row_major);
    __syncthreads();

    for (int i = laneId; i < WM*WN; i += 32) {
        int r = i / WN;
        int c = i % WN;
        int gr = tileM + r;
        int gc = tileN + c;

        if (gr < M && gc < N) {
            float ab = sC[r*WN + c];
            float cold = __half2float(C[gr*N + gc]);
            C[gr*N + gc] = __float2half_rn(alpha * ab + beta * cold);
        }
    }
}

// A, B, and C are device pointers
extern "C" void solve(
    const half* A, const half* B, half* C, int M, int N, int K, float alpha, float beta) {
    dim3 grid((M+WM-1)/WM, (N+WN-1)/WN);
    dim3 block(32, 1, 1);
    size_t shmem_bytes = (WM*WK + WK*WN) * sizeof(half) + (WM*WN) * sizeof(float);
    gemm_kernel<<<grid, block, shmem_bytes>>>(A, B, C, M, N, K, alpha, beta);
    hipDeviceSynchronize();
}
