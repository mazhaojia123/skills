#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <vector>
#include "hip/hip_runtime_api.h"

#define Clock std::chrono::high_resolution_clock

__global__ void addKernel(int *c, const int *a, const int *b, int n)
{
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    c[gtid] = a[gtid] + b[gtid];
}

int main() {
    const int arraySize = 5;
	int size = arraySize;

    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    int threads = 256;
    int blocks = (arraySize + threads - 1) / threads;

    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // For Graph
    hipStream_t streamForGraph;
    hipGraph_t graph;
    std::vector<hipGraphNode_t> nodeDependencies;
    hipGraphNode_t memcpyNode, kernelNode;
    hipKernelNodeParams kernelNodeParams = { 0 };
    hipMemcpy3DParms memcpyParams = { 0 };


	// NOTE: 分配 device memory
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	cudaStatus = hipMalloc((void**)&dev_c, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	// NOTE: 创建图
    checkCudaErrors(hipGraphCreate(&graph, 0));
    checkCudaErrors(hipStreamCreateWithFlags(&streamForGraph, hipStreamNonBlocking));

	// NOTE: 添加 memcpy 节点
    // Add memcpy nodes for copying input vectors from host memory to GPU buffers
    memset(&memcpyParams, 0, sizeof(memcpyParams));
    memcpyParams.srcArray = NULL;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = make_hipPitchedPtr((void*)a, size * sizeof(int), size, 1);
    memcpyParams.dstArray = NULL;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = make_hipPitchedPtr(dev_a, size * sizeof(float), size, 1);
    memcpyParams.extent = make_hipExtent(size * sizeof(float), 1, 1);
    memcpyParams.kind = hipMemcpyHostToDevice;
    checkCudaErrors(hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams));
    nodeDependencies.push_back(memcpyNode);

    memset(&memcpyParams, 0, sizeof(memcpyParams));
    memcpyParams.srcArray = NULL;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = make_hipPitchedPtr((void*)b, size * sizeof(int), size, 1);
    memcpyParams.dstArray = NULL;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = make_hipPitchedPtr(dev_b, size * sizeof(float), size, 1);
    memcpyParams.extent = make_hipExtent(size * sizeof(float), 1, 1);
    memcpyParams.kind = hipMemcpyHostToDevice;
    checkCudaErrors(hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams));
    nodeDependencies.push_back(memcpyNode);

	// NOTE: 添加 kenrel 节点
    // Add a kernel node for launching a kernel on the GPU
    memset(&kernelNodeParams, 0, sizeof(kernelNodeParams));
    kernelNodeParams.func = (void*)addKernel;
    kernelNodeParams.gridDim = dim3(blocks, 1, 1);
    kernelNodeParams.blockDim = dim3(threads, 1, 1);
    kernelNodeParams.sharedMemBytes = 0;
    void* kernelArgs[4] = { (void*)&dev_c, (void*)&dev_a, (void*)&dev_b, &size };
    kernelNodeParams.kernelParams = kernelArgs;
    kernelNodeParams.extra = NULL;
    checkCudaErrors(hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(), nodeDependencies.size(), &kernelNodeParams));
    nodeDependencies.clear();
    nodeDependencies.push_back(kernelNode);

	// NOTE: 添加 memcpy 节点
    // Add memcpy node for copying output vector from GPU buffers to host memory
    memset(&memcpyParams, 0, sizeof(memcpyParams));
    memcpyParams.srcArray = NULL;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = make_hipPitchedPtr(dev_c, size * sizeof(int), size, 1);
    memcpyParams.dstArray = NULL;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = make_hipPitchedPtr(c, size * sizeof(int), size, 1);
    memcpyParams.extent = make_hipExtent(size * sizeof(int), 1, 1);
    memcpyParams.kind = hipMemcpyDeviceToHost;
    checkCudaErrors(hipGraphAddMemcpyNode(&memcpyNode, graph, nodeDependencies.data(), nodeDependencies.size(), &memcpyParams)); 

	// NOTE: 输出有用的信息
	hipGraphNode_t* nodes = NULL;
	size_t numNodes = 0;
	checkCudaErrors(hipGraphGetNodes(graph, nodes, &numNodes));
	printf("Num of nodes in the graph created manually = %zu\n", numNodes);

	// NOTE: 初始化图
    // Create an executable graph from a graph
    hipGraphExec_t graphExec;
    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
    auto t1 = Clock::now();
    for (int i = 0; i < 100; ++i) {
		// NOTE: 在某个流上执行图
        checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
		// NOTE: 同步流
        checkCudaErrors(hipStreamSynchronize(streamForGraph));
    }
    auto t2 = Clock::now();
    auto us_elapsed = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    printf("Looped %d time(s) in %lld microseconds\n", 100, us_elapsed.count());

	printf("C:\t");
	for (int i = 0; i < arraySize; i++){
		printf("%d\t", c[i]);
	}
	printf("\n");

    // Clean up
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipStreamDestroy(streamForGraph));

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}